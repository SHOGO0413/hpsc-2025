
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>

__global__ void count_kernel(int* key, int* bucket, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        atomicAdd(&bucket[key[idx]], 1);
    }
}

// prefix sum（シリアル。少数rangeならOK）
void prefix_sum_cpu(int* bucket, int* prefix, int range) {
    prefix[0] = 0;
    for (int i = 1; i < range; i++) {
        prefix[i] = prefix[i - 1] + bucket[i - 1];
    }
}

// GPUでkey配列を再構築
__global__ void rebuild_kernel(int* sorted_key, int* prefix, int* bucket, int range) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < range && bucket[idx] > 0) {
        int start = prefix[idx];
        for (int i = 0; i < bucket[idx]; i++) {
            sorted_key[start + i] = idx;
        }
    }
}

int main() {
    int n = 50;
    int range = 5;

    std::vector<int> key(n);
    for (int i = 0; i < n; i++) {
        key[i] = rand() % range;
        printf("%d ", key[i]);
    }
    printf("\n");

    // GPUメモリ確保
    int *d_key, *d_bucket, *d_sorted;
    hipMalloc(&d_key, n * sizeof(int));
    hipMalloc(&d_bucket, range * sizeof(int));
    hipMalloc(&d_sorted, n * sizeof(int));
    hipMemset(d_bucket, 0, range * sizeof(int));

    hipMemcpy(d_key, key.data(), n * sizeof(int), hipMemcpyHostToDevice);

    // カウントカーネル呼び出し
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    count_kernel<<<blocks, threads>>>(d_key, d_bucket, n);
    hipDeviceSynchronize();

    // カウント結果をCPUにコピー
    std::vector<int> bucket(range);
    hipMemcpy(bucket.data(), d_bucket, range * sizeof(int), hipMemcpyDeviceToHost);

    // prefix sum（CPUで処理）
    std::vector<int> prefix(range);
    prefix_sum_cpu(bucket.data(), prefix.data(), range);

    // prefixとbucketをGPUに送る
    int *d_prefix;
    hipMalloc(&d_prefix, range * sizeof(int));
    hipMemcpy(d_prefix, prefix.data(), range * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_bucket, bucket.data(), range * sizeof(int), hipMemcpyHostToDevice);

    // 再構築カーネル呼び出し
    rebuild_kernel<<<1, range>>>(d_sorted, d_prefix, d_bucket, range);
    hipDeviceSynchronize();

    // 結果をCPUにコピー
    hipMemcpy(key.data(), d_sorted, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d ", key[i]);
    }
    printf("\n");

    // メモリ解放
    hipFree(d_key);
    hipFree(d_bucket);
    hipFree(d_sorted);
    hipFree(d_prefix);

    return 0;
}
